//Nesma Belkhodja
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>

/**
This method finds the max in the array by splitting the array in two and 
comparing the integers. By the end of the procedure, the greatest int will be the first
element of the results array.
*/
__global__ void getmaxcu(int* memNums, int* memResults, int size) {
    unsigned int t_id = threadIdx.x;
    int numActiveThreads = blockDim.x;
    int num = (numActiveThreads * blockIdx.x) + t_id;
    if (num < size) {
        atomicMax((int*)memResults, (int)memNums[num]);
    }
    else {
        int result[1024];
        result[t_id] = memNums[num];
        int half;
        while(numActiveThreads > 32){
            half = numActiveThreads/2;

            if (t_id < half){
                int temp = result[t_id + half];
                if (temp > result[t_id]){
                    result[t_id] = temp;
                }
            }
        numActiveThreads = numActiveThreads/2;
        }
    }

}

/**
This method handles the memory and kernal call. It first copies over the memory,
makes call to kernal, copies over the results, and then frees the memory.
It returns the max int in the array.
*/
int memoryAndKernalCalls(unsigned int * numbers, unsigned int * result, int size){ 
    int * memNums;
    int * memResults;
    int intSize = sizeof(int);
    int sizeSq = size*intSize;
    int blocks = ceil(size/(double)1024);

    hipMalloc((void **)&memNums, sizeSq);
    hipMalloc((void **)&memResults, intSize);

    hipMemcpy(memNums, numbers, sizeSq, hipMemcpyHostToDevice);
    hipMemcpy(memResults, result, intSize, hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    getmaxcu<<<blocks, 1024>>>(memNums, memResults, size); //1024 threads per block

    hipMemcpy(result, memResults, intSize, hipMemcpyDeviceToHost);
    hipSetDevice(1);
    
    hipFree(memNums);
    hipFree(memResults);
    hipDeviceReset();

    int max = result[0];
    return(max);
}

/**
Main method, much like that of maxseq.c code provided, reads input, checks for
errors, populates the array with random numbers, and calls the appropriate 
methods to find the largest integer in 
the array.
*/
int main(int argc, char *argv[])
{
    double total_time;
    clock_t start, end;
    start = clock();
    unsigned int size = 0;  // The size of the array
    unsigned int i;  // loop index
    unsigned int * numbers; //pointer to the array
    unsigned int * result;
    
    if(argc !=2)
    {
       printf("usage: maxseq num\n");
       printf("num = size of the array\n");
       exit(1);
    }
   
    size = atol(argv[1]);
    result = (unsigned int *)malloc(sizeof(unsigned int));
    result[0] = 0;
    numbers = (unsigned int *)malloc(size * sizeof(unsigned int *));
    
    if( !numbers )
    {
       printf("Unable to allocate mem for an array of size %ld\n", size);
       exit(1);
    }

    srand(time(NULL)); // setting a seed for the random number generator
    // Fill-up the array with random numbers from 0 to size-1 
    for( i = 0; i < size; i++)
       numbers[i] = rand() % size;  
    
    int max = memoryAndKernalCalls(numbers, result, size);
    
    // print the result
    printf("The maximum number in the array is: %ld\n", max);

    free(numbers);

    //find end time
    end = clock(); //time count stops 
    total_time = ((double) (end - start)) / CLOCKS_PER_SEC;
    printf("\nTime taken: %f\n", total_time); //note: this is not the time used for computing results 
    //(the Linux time command was used) this was just for debugging purposes

    exit(0);
}

/*
   input: pointer to an array of long int
          number of elements in the array
   output: the maximum number of the array
*/
int getmax(unsigned int num[], unsigned int size)
{
  unsigned int i;
  unsigned int max = num[0];

  for(i = 1; i < size; i++)
    if(num[i] > max)
       max = num[i];

  return( max );

}